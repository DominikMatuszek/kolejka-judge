
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdexcept>

__device__ bool is_prime(uint64_t number) {
    if ( number < 2 )
        return false;
    if ( number == 2 )
        return true;
    if ( number % 2 == 0 )
        return false;
    for ( uint64_t i = 3 ; i*i <= number ; i+=2 ) {
        if ( number % i == 0 )
            return false;
    }
    return true;
}

__global__ void run(unsigned num_cases, uint64_t* values, char* results) {
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx < num_cases )
        results[idx] = is_prime(values[idx]) ? 1 : 0;
}

void check_cuda(hipError_t err) {
    if (err != hipSuccess) 
        throw std::runtime_error(hipGetErrorString(err));
}

int main() {
    unsigned num_cases;
    std::cin >> num_cases;
    
    uint64_t* values = reinterpret_cast<uint64_t*>(malloc(num_cases*sizeof(uint64_t)));
    uint64_t* d_values;
    check_cuda(hipMalloc(&d_values, num_cases*sizeof(uint64_t)));
    char* results = reinterpret_cast<char*>(malloc(num_cases));
    char* d_results;
    check_cuda(hipMalloc(&d_results, num_cases*sizeof(char)));

    for ( unsigned c = 0 ; c < num_cases ; c++ )
        std::cin >> values[c];

    check_cuda(hipMemcpy(d_values, values, num_cases*sizeof(uint64_t), hipMemcpyHostToDevice));
    run<<<(num_cases+255)/256, 256>>>(num_cases, d_values, d_results);
    check_cuda(hipMemcpy(results, d_results, num_cases*sizeof(char), hipMemcpyDeviceToHost));

    for ( unsigned c = 0 ; c < num_cases ; c++ ) {
        if ( results[c] == 1 )
            std::cout << "yes" << std::endl;
        else
            std::cout << "no " << std::endl;
    }

    free(values);
    check_cuda(hipFree(d_values));
    free(results);
    check_cuda(hipFree(d_results));

    return 0;
}
